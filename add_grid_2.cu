#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>


__global__ void init(int n, float *x, float* y) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
}

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y) {
    // threadIdx.x contains the index of the current thread within its block,
    // blockDim.x contains the number of threads in the block
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void) {
    int N = 1 << 20;
    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the device

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    init << < numBlocks, blockSize >> > (N, x, y);
    add << < numBlocks, blockSize >> > (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}